#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

#define NX 10
#define NY 10
#define INF 100
#define F 1.0
#define BLOCK_SIZE 4
#define MAX_BLOCKS_X ((NX + BLOCK_SIZE - 1) / BLOCK_SIZE)
#define MAX_BLOCKS_Y ((NY + BLOCK_SIZE - 1) / BLOCK_SIZE)
#define MAX_BLOCKS (MAX_BLOCKS_X * MAX_BLOCKS_Y)
#define THREADS_PER_BLOCK 1024

__device__ __host__ inline int getIndex(int bx, int by) {
    return bx * MAX_BLOCKS_Y + by;
}

struct Subdomain {
    double data[BLOCK_SIZE + 2][BLOCK_SIZE + 2];
    int CL;
};

__device__ double computeSDDevice(Subdomain* sd) {
    double sum = 0.0;
    for (int i = 1; i <= BLOCK_SIZE; i++) {
        for (int j = 1; j <= BLOCK_SIZE; j++) {
            sum += sd->data[i][j];
        }
    }
    return sum / (BLOCK_SIZE * BLOCK_SIZE);
}

__device__ void ComputeSubdomain(Subdomain* sd, int bx, int by, double* SD) {
    for (int i = 1; i <= BLOCK_SIZE; i++) {
        for (int j = 1; j <= BLOCK_SIZE; j++) {
            double tx = fmin(sd->data[i - 1][j], sd->data[i + 1][j]);
            double ty = fmin(sd->data[i][j - 1], sd->data[i][j + 1]);
            double diff = fabs(tx - ty);

            double updated;
            if (diff >= 1.0 / F)
                updated = fmin(tx, ty) + 1.0 / F;
            else
                updated = (tx + ty + sqrt(2.0 / (F * F) - diff * diff)) / 2.0;

            if (updated < sd->data[i][j])
                sd->data[i][j] = updated;
        }
    }
    SD[getIndex(bx, by)] = computeSDDevice(sd);
}

__global__ void ComputeScheduleKernel(Subdomain* d_blocks, double* d_SD, int* d_schedule, int* d_CL, int noSched, int* d_noActive) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid >= noSched) return;

    int id = d_schedule[tid];
    int bx = id / MAX_BLOCKS_Y;
    int by = id % MAX_BLOCKS_Y;

    if (d_CL[id] == 1) {
        ComputeSubdomain(&d_blocks[id], bx, by, d_SD);
        d_CL[id] = 0;
        atomicAdd(d_noActive, 1);
    }
}

__global__ void syncGhostCellsKernel(Subdomain* d_blocks, int* d_schedule, int noSched) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid >= noSched) return;

    int id = d_schedule[tid];
    int bx = id / MAX_BLOCKS_Y;
    int by = id % MAX_BLOCKS_Y;

    for (int dir = -1; dir <= 1; dir += 2) {
        int nbx = bx + dir;
        if (nbx >= 0 && nbx < MAX_BLOCKS_X) {
            for (int j = 1; j <= BLOCK_SIZE; j++) {
                int idx1 = getIndex(bx, by);
                int idx2 = getIndex(nbx, by);
                d_blocks[idx2].data[(dir == -1 ? BLOCK_SIZE + 1 : 0)][j] =
                    d_blocks[idx1].data[(dir == -1 ? 1 : BLOCK_SIZE)][j];
            }
        }
        int nby = by + dir;
        if (nby >= 0 && nby < MAX_BLOCKS_Y) {
            for (int i = 1; i <= BLOCK_SIZE; i++) {
                int idx1 = getIndex(bx, by);
                int idx2 = getIndex(bx, nby);
                d_blocks[idx2].data[i][(dir == -1 ? BLOCK_SIZE + 1 : 0)] =
                    d_blocks[idx1].data[i][(dir == -1 ? 1 : BLOCK_SIZE)];
            }
        }
    }
}

Subdomain* h_blocks;
double* h_SD;
int h_CL[MAX_BLOCKS];
int h_schedule[MAX_BLOCKS];

double oldAv = INF;
int noC = 1;

void InitializeProblem() {
    h_blocks = (Subdomain*)malloc(MAX_BLOCKS * sizeof(Subdomain));
    h_SD = (double*)malloc(MAX_BLOCKS * sizeof(double));

    for (int bx = 0; bx < MAX_BLOCKS_X; bx++) {
        for (int by = 0; by < MAX_BLOCKS_Y; by++) {
            int id = getIndex(bx, by);
            Subdomain* sd = &h_blocks[id];
            for (int i = 0; i < BLOCK_SIZE + 2; i++) {
                for (int j = 0; j < BLOCK_SIZE + 2; j++) {
                    sd->data[i][j] = INF;
                }
            }
            sd->CL = 0;
            h_SD[id] = INF;
        }
    }

    int cx = NX / 2;
    int cy = NY / 2;
    int bx = cx / BLOCK_SIZE;
    int by = cy / BLOCK_SIZE;
    int lx = (cx % BLOCK_SIZE) + 1;
    int ly = (cy % BLOCK_SIZE) + 1;
    int id = getIndex(bx, by);
    h_blocks[id].data[lx][ly] = 0.0;
    h_blocks[id].CL = 1;
    h_SD[id] = 0.0;
    h_CL[id] = 1;
}

int BuildSchedule(int* schedule) {
    int noActive = 0;
    double sumSD = 0.0;
    double cutT = INF;

    for (int id = 0; id < MAX_BLOCKS; id++) {
        if (h_CL[id] == 1) {
            noActive++;
            sumSD += h_SD[id];
        }
    }

    if (noActive > 2 * noC) {
        double Av = sumSD / noActive;
        cutT = Av + 0.4 * fmax(0.0, Av - oldAv);
        oldAv = Av;
    }

    int noSched = 0;
    for (int id = 0; id < MAX_BLOCKS; id++) {
        if (h_CL[id] == 1 && h_SD[id] < cutT) {
            schedule[noSched++] = id;
        }
    }
    return noSched;
}

void PrintGlobalGrid() {
    for (int gx = 0; gx < NX; gx++) {
        for (int gy = 0; gy < NY; gy++) {
            int bx = gx / BLOCK_SIZE;
            int by = gy / BLOCK_SIZE;
            int lx = (gx % BLOCK_SIZE) + 1;
            int ly = (gy % BLOCK_SIZE) + 1;
            int id = getIndex(bx, by);
            printf("%7.2f ", h_blocks[id].data[lx][ly]);
        }
        printf("\n");
    }
}

int main() {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    noC = prop.multiProcessorCount * prop.maxThreadsPerMultiProcessor;
    printf("GPU supports %d active threads across %d SMs. noC set to %d\n", prop.maxThreadsPerMultiProcessor, prop.multiProcessorCount, noC);

    InitializeProblem();

    Subdomain* d_blocks;
    double* d_SD;
    int* d_schedule;
    int* d_CL;
    int* d_noActive;

    hipMalloc(&d_blocks, MAX_BLOCKS * sizeof(Subdomain));
    hipMalloc(&d_SD, MAX_BLOCKS * sizeof(double));
    hipMalloc(&d_schedule, MAX_BLOCKS * sizeof(int));
    hipMalloc(&d_CL, MAX_BLOCKS * sizeof(int));
    hipMalloc(&d_noActive, sizeof(int));

    hipMemcpy(d_blocks, h_blocks, MAX_BLOCKS * sizeof(Subdomain), hipMemcpyHostToDevice);
    hipMemcpy(d_SD, h_SD, MAX_BLOCKS * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_CL, h_CL, MAX_BLOCKS * sizeof(int), hipMemcpyHostToDevice);

    // time execution cal
    hipEvent_t start, stop;
    float elapsedTime;

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    //SolverSOLAS

    int noSched = BuildSchedule(h_schedule);
    int noActive = noSched;

    while (noSched > 0) {
        while ((double)noActive / noSched > 1.0 / 64.0) {
            hipMemcpy(d_schedule, h_schedule, noSched * sizeof(int), hipMemcpyHostToDevice);
            hipMemset(d_noActive, 0, sizeof(int));

            int blocks = (noSched + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
            ComputeScheduleKernel << <blocks, THREADS_PER_BLOCK >> > (d_blocks, d_SD, d_schedule, d_CL, noSched, d_noActive);
            hipDeviceSynchronize();

            syncGhostCellsKernel << <blocks, THREADS_PER_BLOCK >> > (d_blocks, d_schedule, noSched);
            hipDeviceSynchronize();

            hipMemcpy(h_blocks, d_blocks, MAX_BLOCKS * sizeof(Subdomain), hipMemcpyDeviceToHost);
            hipMemcpy(h_SD, d_SD, MAX_BLOCKS * sizeof(double), hipMemcpyDeviceToHost);
            hipMemcpy(h_CL, d_CL, MAX_BLOCKS * sizeof(int), hipMemcpyDeviceToHost);
            hipMemcpy(&noActive, d_noActive, sizeof(int), hipMemcpyDeviceToHost);

            if (noActive < noC) noActive = 0;
        }
        noSched = BuildSchedule(h_schedule);
        noActive = noSched;
    }
    //
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);

    printf("Execution time: %.2f ms \n", elapsedTime);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    //

    printf("Final travel time grid (CUDA SOLAS):\n");
    PrintGlobalGrid();

    hipFree(d_blocks);
    hipFree(d_SD);
    hipFree(d_schedule);
    hipFree(d_CL);
    hipFree(d_noActive);
    free(h_blocks);
    free(h_SD);
    return 0;
}